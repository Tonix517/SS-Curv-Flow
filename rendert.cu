#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <algorithm>

#include "GL/glee.h"
#include "GL/glui.h"
#include "GL/glut.h"

#include "global.h"
#include "consts.h"
#include "particle.h"
#include "math_util.h"
#include "shader.h"

///
///		Render SkyBox
	
float fSkyFaceDim = 600;

void renderSkyBox()
{
	glEnable(GL_TEXTURE_2D);

	//	Ground
	glBindTexture(GL_TEXTURE_2D, skyTex[4]);
	float groundCtr[3] = {CamPos[0], CamPos[1] - fSkyFaceDim / 2.f, CamPos[2]};
	glBegin(GL_QUADS);
		glTexCoord2d(0, 0);	glVertex3f(groundCtr[0] - fSkyFaceDim / 2.f, groundCtr[1], groundCtr[2] - fSkyFaceDim / 2.f);
		glTexCoord2d(0, 1);	glVertex3f(groundCtr[0] - fSkyFaceDim / 2.f, groundCtr[1], groundCtr[2] + fSkyFaceDim / 2.f);
		glTexCoord2d(1, 1);	glVertex3f(groundCtr[0] + fSkyFaceDim / 2.f, groundCtr[1], groundCtr[2] + fSkyFaceDim / 2.f);
		glTexCoord2d(1, 0);	glVertex3f(groundCtr[0] + fSkyFaceDim / 2.f, groundCtr[1], groundCtr[2] - fSkyFaceDim / 2.f);
	glEnd();

	//	X-
	glBindTexture(GL_TEXTURE_2D, skyTex[0]);
	float x_m_ctr[3] = {CamPos[0] - fSkyFaceDim / 2.f, CamPos[1], CamPos[2]};
	glBegin(GL_QUADS);
		glTexCoord2d(1, 1);	glVertex3f(x_m_ctr[0], x_m_ctr[1] - fSkyFaceDim / 2.f, x_m_ctr[2] - fSkyFaceDim / 2.f);
		glTexCoord2d(1, 0);	glVertex3f(x_m_ctr[0], x_m_ctr[1] + fSkyFaceDim / 2.f, x_m_ctr[2] - fSkyFaceDim / 2.f);
		glTexCoord2d(0, 0);	glVertex3f(x_m_ctr[0], x_m_ctr[1] + fSkyFaceDim / 2.f, x_m_ctr[2] + fSkyFaceDim / 2.f);
		glTexCoord2d(0, 1);	glVertex3f(x_m_ctr[0], x_m_ctr[1] - fSkyFaceDim / 2.f, x_m_ctr[2] + fSkyFaceDim / 2.f);
	glEnd();

	//	-Z
	glBindTexture(GL_TEXTURE_2D, skyTex[1]);
	float z_m_ctr[3] = { CamPos[0], CamPos[1], CamPos[2] - fSkyFaceDim / 2.f};
	glBegin(GL_QUADS);
		glTexCoord2d(0, 1);	glVertex3f(z_m_ctr[0] - fSkyFaceDim / 2.f, z_m_ctr[1] - fSkyFaceDim / 2.f, z_m_ctr[2]);
		glTexCoord2d(1, 1);	glVertex3f(z_m_ctr[0] + fSkyFaceDim / 2.f, z_m_ctr[1] - fSkyFaceDim / 2.f, z_m_ctr[2]);
		glTexCoord2d(1, 0);	glVertex3f(z_m_ctr[0] + fSkyFaceDim / 2.f, z_m_ctr[1] + fSkyFaceDim / 2.f, z_m_ctr[2]);
		glTexCoord2d(0, 0);	glVertex3f(z_m_ctr[0] - fSkyFaceDim / 2.f, z_m_ctr[1] + fSkyFaceDim / 2.f, z_m_ctr[2]);
	glEnd();

}

///

void resize(int w, int h)
{
	glViewport(0, 0, (GLsizei) w, (GLsizei) h);
}

static 
void destroy()
{	
	global_destroy();

	math_destroy();
	//
	unlinkShaders();
	unloadCubemapTex();

	unloadSkyBoxTex();

	//	DevIL finalization
	ilDeleteImages(1, &nCurrImg);

	exit(EXIT_SUCCESS);
}

static void renderBitmapString(float x, float y, void *font, char *string) 
{  
	char *c;
	glRasterPos2f(x,y);

	for (c = string; *c != '\0'; c++) 
	{
		glutBitmapCharacter(font, *c);
	}
}

static void printFPS()
{
	static clock_t nLastTick = 0;

	glDisable(GL_LIGHTING);
	glDisable(GL_TEXTURE_2D);

	glMatrixMode(GL_PROJECTION);
	glPushMatrix();
	glLoadIdentity();

	gluOrtho2D(0, 1, 0, 1);

	glColor3f(1, 1, 0);

	glUseProgram(0);

	//	Calculate FPS
	clock_t nCurrTick = clock();

	char buf[20] = {0};
	sprintf(buf, "%.2f fps", 1000.f / (nCurrTick - nLastTick));

	glColor3f(1, 1, 0);
	renderBitmapString(0, 0.95, GLUT_BITMAP_TIMES_ROMAN_24, buf);
	
	glPopMatrix();

	nLastTick = nCurrTick;

	glEnable(GL_TEXTURE_2D);
	glEnable(GL_LIGHTING);

	glUseProgram(program);
}

void display()
{

	clock_t t0 = clock();

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glBindTexture(GL_TEXTURE_2D, 0);

	{
		//	Pass CamPos as eyePos into V-Shader
		GLint eyePos;
		eyePos = glGetUniformLocation(program, "eyepos");
		glUniform3f(eyePos, CamPos[0], CamPos[1], CamPos[2]);

		GLint fluidColorInx = glGetUniformLocation(program, "fluidColor");
		glUniform4f(fluidColorInx, 0.1, 0.3, 0.6, 1);

		///
		///		Set View
		///
		glMatrixMode(GL_PROJECTION);
		glPushMatrix();
		glLoadIdentity();
	    	
		float fScope = 0.5;
		glFrustum(-fScope, fScope, -fScope, fScope, 1, 1000);
		gluLookAt( CamPos[0], CamPos[1], CamPos[2],
				   CamTarget[3], CamTarget[1], CamTarget[2],
				   CamUp[0], CamUp[1], CamUp[2]);

		glTranslatef(InitDist, InitDist * 0.8, InitDist);
	
		{
			glGetDoublev(GL_MODELVIEW_MATRIX, modelview);
			glGetDoublev(GL_PROJECTION_MATRIX, projection);
			glGetIntegerv( GL_VIEWPORT, viewport );
		}

		///
		///		Render
		///
		resetGrid();

		//	Render Particles
		for(int i = 0; i< particles.size(); i ++)
		{
			particles[i].render();
		}

clock_t t1 = clock();

		//	get depth buffer of Particles
		//
		glFlush();
		glReadPixels(0, 0, width, height, GL_DEPTH_COMPONENT, GL_FLOAT, depth_buffer);

		///
		///		render SkyBox
		///
		{
			glUseProgram(0);
				
			/* Lighting Variables */
			GLfloat light_ambient[] = { 1.0, 1.0, 1.0, 1.0 }; 
			GLfloat light_diffuse[] = { 1.0, 1.0, 1.0, 1.0 };
			GLfloat light_specular[] = { 1.0, 1.0, 1.0, 1.0 };
			GLfloat light_position[] = { 1.0, 1.0, 1.0, 0.0 };

			glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
			glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
			glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
			glLightfv(GL_LIGHT0, GL_POSITION, light_position);

			glEnable(GL_LIGHTING);
			glEnable(GL_LIGHT0);
				renderSkyBox();				
			glDisable(GL_LIGHT0);
			glDisable(GL_LIGHTING);
			
			glUseProgram(program);
		}

		//	get depth buffer of the whole scene
		//
		glFlush();
		glReadPixels(0, 0, width, height, GL_DEPTH_COMPONENT, GL_FLOAT, depth_buffer1);

		glPopMatrix();
		
		//
		//	Processing Z-Buffer data		
		//
		genDepthPic(depth_buffer, depth_buffer1, depth_buffer, width, height);

clock_t t2 = clock();	

		//	Thickness
		projectAllParticles(particles, particles2d);
		calcAllThickness(thick_buffer, depth_buffer, width, height, particles2d);

clock_t t3 = clock();		

		//	Curvature Flow
		memset(cf_buf, 0, sizeof(float) * WinWidth * WinHeight);
		curvature_flow(depth_buffer, fCFFactor, width, height, nIterCount);

clock_t t4 = clock();		

		////	Normals
		calcAllNormals(normals, depth_buffer, width, height, nNormalsLerpStep);

clock_t t5 = clock();

		///
		///		Render using Particles
		///
		glMatrixMode(GL_PROJECTION);	
		glPushMatrix();

		glLoadIdentity();
		gluOrtho2D(0, 1, 0, 1);

		//	Fluid vertices
		//
		glUseProgram(program);

		glColor3f(1, 0, 0);
		glBegin(GL_POINTS);
		for(int j = 0; j < height; j ++)
		for(int i = 0; i < width; i ++)
		{
			float fd = *(depth_buffer + (i + j * width));
			if(fd > 0)
			{					
				glVertexAttrib1f(thickAttribInx, *(thick_buffer + (i + j * width)));
				glNormal3fv(normals + (i + j * width) * 3);
				glVertex2f(i * 1.f/ width, j * 1.f/ height);				
			}
		}
		glEnd();

		glPopMatrix();

clock_t t6 = clock();

		for(int i = 0; i < particles.size(); i ++)
		{
			particles[i].update();
		}
		
		clock_t t7 = clock();	
		clock_t nTotal = t7 - t0;
		printf(" - IO : %.2f, Thick : %.2f, CF : %.2f, Norm : %.2f, Vertex : %.2f, Physics : %.2f \n", 
				(t2 - t1) * 1.f / nTotal, (t3 - t2) * 1.f / nTotal, (t4 - t3) * 1.f / nTotal, 
				(t5 - t4) * 1.f / nTotal, (t6 - t5) * 1.f / nTotal, (t7 - t6) * 1.f / nTotal  );

	}	

	printFPS();

	glutSwapBuffers();
}

static 
void key(unsigned char key, int x, int y)
{		
    switch (key) 
    {
	case 'c':
	case 'C':
		//	Taking & Saving the screenshot				   
		if(ilutGLScreen())
		{					  
		  ilEnable(IL_FILE_OVERWRITE);
		  char buf[30] = {0};
		  sprintf(buf, "fl_%d.jpg", nCurrImgCount++);
		  if(ilSaveImage(buf))
		  {
			 printf("Screenshot saved successfully as \'%s\'!\n",buf);
		  }
		  else
		  {
			 printf("Sorry, DevIL cannot save your screenshot...\n");
		  }
		}
		else
		{
		  printf(" Sorry man, DevIL screenshot taking failed...\n");
		}
		break;

    case 27 : 
    case 'q':
        destroy();
        break;
    }

    glutPostRedisplay();
}

int iWinId;
void idle() 
{
	glutSetWindow(iWinId);
	glutPostRedisplay();
}

///
void callback_gpu_enable(int)
{

}

///	Reset the scene...
void callback_reset(int)
{
	global_destroy();
	global_init();

	particles.clear();
	Particle::genParticlesByRandom(ParCount, clock());
}

///
int main(int argc, char* argv[])
{
	//	Print Usage
	printf("-------------------\n");
	printf(" SPH by Tony Zhang\n");
	printf("-------------------\n");
	printf("Press 'C' to capture screenshot\n");
	printf("Press ESC to exit\n\n");

	//
	srand(clock());

	//	Window Setup
	glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE | GLUT_DEPTH);	
	
	glutInitWindowSize(WinWidth, WinHeight);
    glutInitWindowPosition(WinLeft, WinTop);
    iWinId = glutCreateWindow(WinTitle);
    
	glutReshapeFunc(resize);
    glutDisplayFunc(display);
    glutKeyboardFunc(key);
	glutIdleFunc(idle);

	//	Warning : these init calling has to be put
	//		      after window is created.

	//	Anti-Aliasing for Lines
	glEnable(GL_LINE_SMOOTH);
	glHint(GL_LINE_SMOOTH_HINT, GL_DONT_CARE);

    glDepthFunc(GL_LESS);
    glEnable(GL_DEPTH_TEST);
	
	glEnable(GL_CULL_FACE);
	glCullFace(GL_BACK);
	
	glClearColor(0.75, 0.75, 0.75, 0);

	//	Particle Init	
	Particle::genParticlesByRandom(ParCount, clock());	
	global_init();

	//	DevIL init
	//
	ilInit();
	ilutRenderer(ILUT_OPENGL);
	ilutEnable(ILUT_OPENGL_CONV);

	//	Cubemap Texture setup
	//
	math_init();

	loadCubemapTex("violentdays_large.jpg");

	loadSkyBoxTex("violentdays_large.jpg", skyTex);

	loadShader("shader/v_shader.txt", VERTEX);
	loadShader("shader/f_shader.txt", FRAGMENT);
	linkShaders();

	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_CUBE_MAP, nCubemapTexId);
	cubeMapInx = glGetUniformLocation(program, "cubemapTex");
	glUniform1i(cubeMapInx, 0);

	ilGenImages(1, &nCurrImg);
	ilBindImage(nCurrImg);	

	///

	//	Put Barrier
	//float ctr[3] = {0, 15, 0};
	//Ball *pBall = new Ball(5, ctr);
	//barVec.push_back(pBall);

	//	GLUI
	GLUI *glui = GLUI_Master.create_glui( "Param Control", 0, WinWidth + 30, 20 );
	
	//	GPU setting
	//
	{
		GLUI_Panel *pPGpu = glui->add_panel("GPU Setup");
		GLUI_Checkbox *pPMGpuChk = glui->add_checkbox_to_panel(pPGpu, "GPU Enabled", &bGPUEnabled, -1, callback_gpu_enable);
	}

	{
		//	Particle Count
		GLUI_Spinner *pParCount = glui->add_spinner("Particle Count", GLUI_SPINNER_INT, &ParCount);
		pParCount->set_float_limits(10, 100000);
		pParCount->set_speed(10);

		//	Particle Radius	
		GLUI_Spinner *pPRad = glui->add_spinner("Particle Radius", GLUI_SPINNER_FLOAT, &ParRad);
		pPRad->set_int_limits(0.1, 9);
		pPRad->set_speed(0.5);

	}

	//	Rigid Body Part
	//
	{
		GLUI_Panel *pRBPal = glui->add_panel("Rigid Body Param");

		//	Velocity Factor
		GLUI_Spinner *pVelFac = glui->add_spinner_to_panel(pRBPal, "Velocity Factor", GLUI_SPINNER_FLOAT, &fVelFactor);
		pVelFac->set_int_limits(0.01, 2);
		pVelFac->set_speed(0.05);

		////	Gravity Factor
		//GLUI_Spinner *pGravFac = glui->add_spinner_to_panel(pRBPal, "Gravity Factor", GLUI_SPINNER_FLOAT, &fGravFactor);
		//pGravFac->set_int_limits(0.01, 2);
		//pGravFac->set_speed(0.05);

		//	Velocity Dissipation by Wall
		GLUI_Spinner *pVelDiss = glui->add_spinner_to_panel(pRBPal, "Vel-Diss by Wall", GLUI_SPINNER_FLOAT, &fVelDissByWall);
		pVelDiss->set_int_limits(0.01, 2);
		pVelDiss->set_speed(0.05);

		//	Rigid Spring Coefficient
		GLUI_Spinner *pRigSpring = glui->add_spinner_to_panel(pRBPal, "Rigid Spring Coefficient", GLUI_SPINNER_FLOAT, &fSpringCoe);
		pRigSpring->set_int_limits(0.01, 50);
		pRigSpring->set_speed(0.1);

		//	Rigid Damping Coefficient
		GLUI_Spinner *pRigDamping = glui->add_spinner_to_panel(pRBPal, "Rigid Damping Coefficient", GLUI_SPINNER_FLOAT, &fDampingCoe);
		pRigDamping->set_int_limits(0.01, 50);
		pRigDamping->set_speed(0.01);
	}
	
	//	Fluid Part
	//
	{
		GLUI_Panel *pFLPal = glui->add_panel("Fluid Param");

		//	Fluid Density Coefficient
		GLUI_Spinner *pDensity = glui->add_spinner_to_panel(pFLPal, "Fluid Density", GLUI_SPINNER_FLOAT, &fStdFluidDensity);
		pDensity->set_int_limits(0.01, 20);
		pDensity->set_speed(0.1);

		//	Fluid Density Factor
		GLUI_Spinner *pDenFactor = glui->add_spinner_to_panel(pFLPal, "Fluid Density Factor", GLUI_SPINNER_FLOAT, &fFluidDensityFactor);
		pDenFactor->set_int_limits(0.01, 20);
		pDenFactor->set_speed(0.1);

		//	Mass per Particle
		GLUI_Spinner *pMass = glui->add_spinner_to_panel(pFLPal, "Mass per Particle", GLUI_SPINNER_FLOAT, &fMassPerPar);
		pMass->set_int_limits(0.01, 20);
		pMass->set_speed(0.1);
		
		////	Grid Count Coefficient
		//GLUI_Spinner *pGridCount = glui->add_spinner_to_panel(pFLPal, "Grid Count Per Dim", GLUI_SPINNER_INT, &GridCountPerDim);
		//pGridCount->set_int_limits(1, 1000);
		//pGridCount->set_speed(2);
		
		//	kPress
		GLUI_Spinner *pPress = glui->add_spinner_to_panel(pFLPal, "Pressure Factor", GLUI_SPINNER_FLOAT, &kPress);
		pPress->set_int_limits(0.0001, 20);
		pPress->set_speed(0.05);

		//	kVisco
		GLUI_Spinner *pVisco = glui->add_spinner_to_panel(pFLPal, "Viscosity Factor", GLUI_SPINNER_FLOAT, &kVisco);
		pVisco->set_int_limits(0.0001, 20);
		pVisco->set_speed(0.05);
	}
	glui->add_button("Reset", 0, callback_reset);

	GLUI_Master.set_glutIdleFunc(idle);

	///
	atexit(destroy);

	//
	glutMainLoop();

	destroy();
	return EXIT_SUCCESS;
}

