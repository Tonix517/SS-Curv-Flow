#include "hip/hip_runtime.h"
#ifndef CURVATURE_FLOW_GPU_CU
#define CURVATURE_FLOW_GPU_CU

#include "vector_gpu.cu"
#include "particle_gpu.cu"

__device__ __constant__ int width  = 0;
__device__ __constant__ int height = 0;
__device__ __constant__ float fCFThreshold  = 0;
__device__ __constant__ float fCVFactor = 0;
__device__ __constant__ float fThickKernRad = 30;
__device__ __constant__ float ParRad = 30;

__device__	float *depth_buf0_gpu = NULL;
__device__  float *avg_normals_gpu;
__device__  float *pixle_normals_gpu;
__device__	Particle_gpu *pars_gpu;
__device__  float *thick_buf_gpu;

void sendConstant2GPU(int iWidth, int iHeight, float ffCFThreshold, float ffCVFactor, float ffThickKernRad, float fParRad)
{
	hipMemcpyToSymbol(HIP_SYMBOL(width), &iWidth, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(height), &iHeight, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(fCFThreshold), &ffCFThreshold, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(fCVFactor), &ffCVFactor, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(fThickKernRad), &ffThickKernRad, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(ParRad), &fParRad, sizeof(float), hipMemcpyHostToDevice);
}

__global__
void gpu_setup(float *pDepBuf0, float *pAvgNorm, float *pPixNorm, Particle_gpu *pParsBuf, float *pThickBuf)
{
	depth_buf0_gpu = pDepBuf0;
	avg_normals_gpu = pAvgNorm;
	pixle_normals_gpu = pPixNorm;
	pars_gpu = pParsBuf;
	thick_buf_gpu = pThickBuf;
}

////

__device__
float dz2x(int x, int y)
{
	if( x < 0 || y < 0 || x >= width || y >= height)
	{
		return 0;
	}

	float v0 = (x <= 0) ? 0 :         *(depth_buf0_gpu + (x - 1 + y * width));
	float v1 =                        *(depth_buf0_gpu + (x + y * width));
	float v2 = (x >= width - 1) ? 0 : *(depth_buf0_gpu + (x + 1 + y * width));

	float ret = 0;

	if( (v0 == 0 && v2 != 0) )
	{
		ret = (v2 - v1);
	}
	else if( (v2 == 0 && v0 != 0) )
	{
		ret = (v1 - v0);
	}
	else
	{
		ret = (  v2 - v0) / 2.f;	
	}

	return ret;
}

__device__
float dz2y(int x, int y)
{
	if( x < 0 || y < 0 || x >= width || y >= height)
	{
		return 0;
	}

	float v0 = (y <= 0) ? 0 :         *(depth_buf0_gpu + (x + (y - 1) * width));
	float v1 =                        *(depth_buf0_gpu + (x + y * width));
	float v2 = (y >= height - 1) ? 0 : *(depth_buf0_gpu + (x + (y + 1) * width));

	float ret = 0;

	if( (v0 == 0 && v2 != 0) )
	{
		ret = (v2 - v1);
	}
	else if( (v2 == 0 && v0 != 0) )
	{
		ret = (v1 - v0);
	}
	else
	{
		ret = ( v2 - v0) / 2.f;	//	TODO
	}

	return ret;
}

__global__
void curvature_flow_step_gpu()
{
	int tid = 0; //	TODO
	int i, j;	//TODO

	float fd = *(depth_buf0_gpu + tid);
	if(fd >= 0)
	{
		float dz_x = dz2x(i, j);
		float dz_x0 = dz2x(i-1, j);
		float dz_x2 = dz2x(i+1, j);
		float dz2x2 = (dz_x2 - dz_x0) / 2.f;

		float dz_y = dz2y(i, j);
		float dz_y0 = dz2y(i, j-1);
		float dz_y2 = dz2y(i, j+1);
		float dz2y2 = (dz_y2 - dz_y0) / 2.f;

		float Cx = i == 0 ? 0 : 2.f / (width * i);	//	TODO ?
		float Cy = j == 0 ? 0 : 2.f / (height * j);	//	TODO ?
		float D = Cy * Cy * dz_x * dz_x + Cx * Cx * dz_y * dz_y + Cx * Cx * Cy * Cy * (1 - fd) * (1 - fd);
		float inv_D32 = 1.f / powf(D, 1.5);

		float ky = 4.f / height / height;
		float kx = 4.f / width / width;
		float dD_x = ky * pow(j, -2.f) * 2 * dz_x * dz2x2 + 
					 kx * dz_y * dz_y * -2 * pow(i, -3.f) + 
					 ky * pow(j, -2.f) * kx * (-2 * pow(i, -3.f) * fd * fd + pow(i, -2.f) * 2 * fd * dz_x);

		float dD_y = kx * pow(i, -2.f) * 2 * dz_y * dz2y2 + 
					 ky * dz_x * dz_x * -2 * pow(j, -3.f) + 
					 kx * pow(i, -2.f) * ky * (-2 * pow(j, -3.f) * fd * fd + pow(j, -2.f) * 2 * fd * dz_y);

		float Ex = 0.5 * dz_x * dD_x - dz2x2 * D;
		float Ey = 0.5 * dz_y * dD_y - dz2y2 * D;

		float fCF = (Cy * Ex + Cx * Ey) * inv_D32 / 2;

		//	Apply
		//
		fCF = fCF > fCFThreshold ? fCFThreshold : fCF;
		*(depth_buf0_gpu + tid) -= fCF * fCVFactor;

		if(*(depth_buf0_gpu + tid) < 0) *(depth_buf0_gpu + tid) = 0;
		if(*(depth_buf0_gpu + tid) > 1) *(depth_buf0_gpu + tid) = 1;
	}
}

////
__device__
void getAvgNormal(float *retNorm, int x, int y, int nSampleRad)
{
	float tmp[3] = {0};
	unsigned nAvailableCount = 0;

	for(int j = -nSampleRad; j <= nSampleRad; j ++)
	for(int i = -nSampleRad; i <= nSampleRad; i ++)
	{
		int currX = x + i;
		int currY = y + j;
		if( currX >=0 && currX < width &&
			currY >=0 && currY < height )
		{
			float fd = *(depth_buf0_gpu + (currX + currY * width));
			if(fd > 0)
			{
				float currNorm[3];

				//	The sampled normal should get an average value ...
				float dz_x = dz2x(currX, currY);
				float dz_y = dz2y(currX, currY);

				float Cx = i == 0 ? 0 : 2.f / (width * currX);	
				float Cy = j == 0 ? 0 : 2.f / (height * currY);	

				float D = Cy * Cy * dz_x * dz_x + Cx * Cx * dz_y * dz_y + Cx * Cx * Cy * Cy * (1 - fd) * (1 - fd);
				if(D == 0)
				{
					continue;
				}
				float rv_sqrtD = 1.f / sqrt(D);

				currNorm[0] = - Cy * dz_x * rv_sqrtD;
				currNorm[1] = - Cx * dz_y * rv_sqrtD;
				currNorm[2] = Cx * Cy * (1 - fd) * rv_sqrtD;

				vecAdd(tmp, currNorm, tmp);
				nAvailableCount ++;
			}
		}
	}

	if(nAvailableCount > 0)
	{
		vecScale(tmp, 1.f/nAvailableCount, tmp);
		normalize(tmp);
		vecCopy(retNorm, tmp);
	}
}

__global__
void calcAvgNormals_gpu(unsigned nSampleStep, unsigned nAvgNormRad)
{
	int tid = 0;	//	TODO
	if(tid >= width * height)
	{
		return;
	}

	int i, j;		//	TODO

	float fd = *(depth_buf0_gpu + tid);
	if(fd > 0 && (i % nSampleStep == 0) && (j % nSampleStep == 0))
	{
		getAvgNormal(avg_normals_gpu + 3 * tid, i, j, nAvgNormRad);
	}
}

__global__
void calcAllNormals_gpu(unsigned nSampleStep)
{
	int tid = 0; // TODO
	if(tid >= width * height)
	{
		return;
	}

	int i, j; // TODO
	if( (i % nSampleStep == 0) && (j % nSampleStep == 0))
	{
		*(pixle_normals_gpu + 3 * tid + 0) = *(avg_normals_gpu + 3 * tid + 0);
		*(pixle_normals_gpu + 3 * tid + 1) = *(avg_normals_gpu + 3 * tid + 1);
		*(pixle_normals_gpu + 3 * tid + 2) = *(avg_normals_gpu + 3 * tid + 2);
	}

	float fd = *(depth_buf0_gpu + tid);
	if(fd > 0)
	{
		//	1. find grid
		int nGridX0 = (i / nSampleStep) * nSampleStep;
		int nGridY0 = (j / nSampleStep) * nSampleStep;

		float fDistPercX = (i - nGridX0) * 1.f / nSampleStep;
		float fDistPercY = (j - nGridY0) * 1.f / nSampleStep;
		
		//	2. Bi-linear Interpolation
		float NullNorm[3] = {0};
		float *currNormals[4];
		currNormals[0] = pixle_normals_gpu + 3 * (nGridX0 + nGridY0 * width);

		if((i + 1) < width && (nGridX0 + nSampleStep) < width ) 
		{
			currNormals[1] = pixle_normals_gpu + 3 * (nGridX0 + nSampleStep + nGridY0 * width);
		}
		else
		{
			currNormals[1] = NullNorm;
		}

		if((j + 1)< height && (nGridY0 + nSampleStep) < height)
		{
			currNormals[2] = pixle_normals_gpu + 3 * (nGridX0 + (nGridY0 + nSampleStep) * width);
		}
		else
		{
			currNormals[2] = NullNorm;
		}

		if(	(i + 1) < width && (j + 1)< height && 
			(nGridY0 + nSampleStep) < height && 
			(nGridX0 + nSampleStep) < width )
		{
			currNormals[3] = pixle_normals_gpu + 3 * (nGridX0 + nSampleStep + (nGridY0 + nSampleStep) * width);
		}
		else
		{
			currNormals[3] = NullNorm;
		}

		float tmp[3], tmp1[3];

		float n01[3] = {0}; 
		vecScale(currNormals[0], (1 - fDistPercX), tmp);
		vecScale(currNormals[1], fDistPercX, tmp1);
		vecAdd(tmp, tmp1, n01);

		float n23[3] = {0};
		vecScale(currNormals[2], (1 - fDistPercX), tmp);
		vecScale(currNormals[3], fDistPercX, tmp1);
		vecAdd(tmp, tmp1, n23);

		float *pCurrNorm = pixle_normals_gpu + 3 * (tid);
		vecScale(n01, (1 - fDistPercY), tmp);
		vecScale(n23, fDistPercY, tmp1);
		vecAdd(tmp, tmp1, pCurrNorm);
	}
}

//////
__device__
void transform_point(float out[4], const float m[16], const float in[4])   
{   
#define M(row,col) m[col*4+row]   
    out[0] =    
        M(0, 0) * in[0] + M(0, 1) * in[1] + M(0, 2) * in[2] + M(0, 3) * in[3];   
    out[1] =   
        M(1, 0) * in[0] + M(1, 1) * in[1] + M(1, 2) * in[2] + M(1, 3) * in[3];   
    out[2] =   
        M(2, 0) * in[0] + M(2, 1) * in[1] + M(2, 2) * in[2] + M(2, 3) * in[3];   
    out[3] =   
        M(3, 0) * in[0] + M(3, 1) * in[1] + M(3, 2) * in[2] + M(3, 3) * in[3];   
#undef M   
}   

__device__ float modelview[16] = {0};					
__device__ float projection[16] = {0}; 
__device__ int viewport[4] = {0};

__device__
bool myProject(float objx, float objy, float objz, 
			   const float  modelMatrix[16], const float projMatrix[16], const int viewport[4], 
			   float *winx, float *winy, float *winz)   
{   
    // matrice transformation   
    float in[4], out[4];   
    //initialize matrice and column vector as a transformer   
    in[0] = objx;   
    in[1] = objy;   
    in[2] = objz;   
    in[3] = 1.0;   
    transform_point(out, modelMatrix, in);  //����ģ����ͼ����   
    transform_point(in, projMatrix, out);   //����ͶӰ����   
    //��������ĵ������Ϊ0   
    if(in[3] == 0.0)   
        return false;   
    //������λ���׼��   
    in[0] /= in[3];   
    in[1] /= in[3];   
    in[2] /= in[3];   
    //�ӿ�����������   
    *winx = viewport[0] + (1 + in[0]) * viewport[2] / 2;   
    *winy = viewport[1] + (1 + in[1]) * viewport[3] / 2;   
    *winz = (1 + in[2]) / 2;   
    return true;   
}  

__global__
void projectPars_gpu(int nCount)
{
	int tid = 0; //	TODO
	if(tid < nCount)
	{
		float vPrjPos[3] = {0};
		Particle_gpu *par = pars_gpu + tid;
		if(!myProject( par->_pos[0], par->_pos[1], par->_pos[2],
						modelview, projection, viewport, 
						vPrjPos + 0, vPrjPos + 1, vPrjPos + 2) )
		{
			return;
		}

		par->x2d = vPrjPos[0];
		par->y2d = vPrjPos[1];
	}
}

#define max(x,y) ( ((x) > (y)) ? (x) : (y))
__device__
float thickKern(int x, int y, float *pos, float fDepth)
{
	float fProjectedSize = ParRad/* * fDepth*/;	// TODO: I know...

	float p1 = fabs(x - pos[0]);
	float p2 = fabs(y - pos[1]);

	if(p1 > fThickKernRad || p2 > fThickKernRad)
	{
		return 0;
	}

	//	Linear here
	return max(0.f, 1.f - fabs(p1/fProjectedSize)/fThickKernRad) * max(0.f, 1.f - fabs(p2/fProjectedSize)/fThickKernRad);
}

__global__
void calcAllDensity()
{
	int tid = 0;	//	TODO
	if(tid < width * height)
	{
		int i, j;	//	TODO
		float fDepth = *(depth_buf0_gpu + tid);
		if(fDepth == 0)
		{
			*(thick_buf_gpu + (tid)) = 0;
		}
		else
		{
			float fThick = 0;
			
			for(int n = 0; n < /*pars2d.size()*/10; n ++)	//	TODO
			{
				fThick += thickKern(i, j, (pars_gpu + n)->_pos, fDepth) * 0.04;
			}

			*(thick_buf_gpu + tid) = fThick;
		}
	}
}

#endif